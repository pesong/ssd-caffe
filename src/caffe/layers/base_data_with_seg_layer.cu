#include <vector>

#include "caffe/layers/base_data_with_seg_layer.hpp"

namespace caffe {

template <typename Dtype>
void BasePrefetchingDataWithSegLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  Batch<Dtype>* batch = prefetch_full_.pop("Data layer prefetch queue empty");

  // pesong LOG(INFO) << "------------------Forward_Gpu-------------------------" ;

  // Reshape to loaded data.
  top[0]->ReshapeLike(batch->data_);
  top[2]->ReshapeLike(batch->label_img_); // added by pesong

  // Copy the data
  caffe_copy(batch->data_.count(), batch->data_.gpu_data(), top[0]->mutable_gpu_data());
  caffe_copy(batch->label_img_.count(), batch->label_img_.gpu_data(), top[2]->mutable_gpu_data()); // added by pesong

  if (this->output_labels_) {
    // Reshape to loaded labels.
    top[1]->ReshapeLike(batch->label_);
    // Copy the labels.
    caffe_copy(batch->label_.count(), batch->label_.gpu_data(),
        top[1]->mutable_gpu_data());
  }
  // Ensure the copy is synchronous wrt the host, so that the next batch isn't
  // copied in meanwhile.
  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
  prefetch_free_.push(batch);
}

INSTANTIATE_LAYER_GPU_FORWARD(BasePrefetchingDataWithSegLayer);

}  // namespace caffe
